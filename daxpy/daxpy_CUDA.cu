#include <iostream>
#include <vector>
#include <chrono>
#include <cmath>
#include <exception>

#include <hip/hip_runtime.h>

// Precondition
__global__
void daxpy(const double a, const double * x, double * y, const std::size_t n) {
  std::size_t i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a*x[i] + y[i];
  }
}

void check_result(const std::vector<double>& result_vector, const double truth_value) {

  for (const double & value : result_vector) {
    if (value != truth_value) {
      throw std::runtime_error("Error: An element in the result vector does not match the truth value.");
    }
  }

}

int main() {

  // Input
  std::size_t vector_size = pow(10,8);
  double a = 2.0;
  double x_initial_value = 1.0;
  double y_initial_value = 1.0;

  // Expected output
  double truth_value = a*x_initial_value + y_initial_value;

  // Initialize vectors on host
  std::vector<double> x(vector_size, x_initial_value);
  std::vector<double> y(vector_size, y_initial_value);

  // Initialize vectors on device
  double* x_device;
  hipMalloc(&x_device, x.size()*sizeof(double));
  hipMemcpy(x_device, x.data(), x.size()*sizeof(double), hipMemcpyHostToDevice);

  double* y_device;
  hipMalloc(&y_device, y.size()*sizeof(double));
  hipMemcpy(y_device, y.data(), y.size()*sizeof(double), hipMemcpyHostToDevice);

  // Set up the clock to report the elapsed time in seconds
  using clock_t = std::chrono::steady_clock;
  using time_point_t = clock_t::time_point;
  using duration_t  = std::chrono::duration<double, std::chrono::seconds::period>;

  int threads_per_block = 256;
  int blocks_per_grid = (x.size() + threads_per_block - 1) / threads_per_block; // int division but rounds up. In case problem size is not evenly divisable by threads_per_block. 

  // Do the math and time it
  time_point_t time_begin = clock_t::now();
  daxpy<<<blocks_per_grid,threads_per_block>>>(a, x_device, y_device, x.size());
  hipDeviceSynchronize(); // Need this so we time the kernel execution time and not just kernel launch time.
  time_point_t time_end = clock_t::now();

  duration_t run_duration = std::chrono::duration_cast<duration_t>(time_end - time_begin);
  double run_time_seconds = run_duration.count();

  hipMemcpy(y.data(), y_device, y.size()*sizeof(double), hipMemcpyDeviceToHost);
  hipFree(x_device);
  hipFree(y_device);

  check_result(y, truth_value);

  std::cout << "vector size = " << vector_size << std::endl;
  std::cout << "run time = " << run_time_seconds << " [s]" << std::endl;

  return 0;
}